
#include <hip/hip_runtime.h>
#include<iostream>
#include<chrono>
using namespace std;
using namespace std::chrono;

#define clock_now high_resolution_clock::now

__global__ void minimum(int *a,int *b,int n)
{
	int block=256*blockIdx.x;
	int mini=7888888;

	for(int i=block;i<min(256+block,n);i++)
	{

		if(mini>a[i])
		{
			mini=a[i];
		}

	}
	b[blockIdx.x]=mini;
}

int find_min(int *a, int n){
    int min = INT_MAX;
    for(int i = 0;i<n;i++){
        if(min > a[i]){
            min = a[i];
        }
    }
    return min;
}
int main() {
    cout << "Enter the size of the array" << endl;
    int n;
    cin >> n;
    //int a[n]; //does not work in some cuda versions
    int *a = (int *)malloc(n * sizeof(int));
    for(int i = 0; i < n; i++) {
        //a[i] = i;
        a[i] = rand();
    }

    auto start = clock_now();
    int x = find_min(a, n);
    auto end = clock_now();

    cout<<"\nThe minimum element in CPU is: "<<x<<endl;
    auto time = end-start;
    cout<<"Time taken: "<<time.count()<<" microseconds";

    int *ad,*bd;
	int size=n*sizeof(int);
    int grids=ceil(n*1.0f/256.0f);

    hipMalloc(&ad,size);
    hipMalloc(&bd,grids*sizeof(int));

    hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
    
    dim3 grid(grids,1);
    dim3 block(1,1);
    
    start = clock_now();

    while(n>1)
	{
		minimum<<<grids,block>>>(ad,bd,n);
		n=ceil(n*1.0f/256.0f);
		hipMemcpy(ad,bd,n*sizeof(int),hipMemcpyDeviceToDevice);
	}

    end = clock_now();

    int ans[2];
	hipMemcpy(ans,ad,4,hipMemcpyDeviceToHost);
    
    cout<<"\nThe minimum element in GPU is: "<<ans[0]<<endl;
    time = end-start;
    cout<<"Time taken: "<<time.count()<<" microseconds";

    
}